#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <cmath>
#include <iostream>
#include <chrono>

// Functor to square the values
struct square {
    __host__ __device__
    double operator()(const double& x) const {
        return x * x;
    }
};

int main() {
    const int n = 10000000;
    double value = 0.0;

    // Host vector
    thrust::host_vector<double> google_host(n);
    for (int i = 0; i < n; i++) {
        std::cin >> value;
        google_host[i] = value;
    }

    // Transfer to device
    thrust::device_vector<double> google_device = google_host;
    thrust::device_vector<double> squared_elements(n);

    auto start = std::chrono::high_resolution_clock::now();

    // Step 1: Square each element
    thrust::transform(google_device.begin(), google_device.end(), squared_elements.begin(), square());

    // Step 2: Sum the squared elements
    double sum_of_squares = thrust::reduce(squared_elements.begin(), squared_elements.end(), 0.0, thrust::plus<double>());

    // Step 3: Compute the magnitude
    double magnitude = std::sqrt(sum_of_squares);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_time = end - start;

    std::cout << "Magnitude (using separate operations): " << magnitude << std::endl;
    std::cout << "Elapsed time: " << elapsed_time.count() << " seconds" << std::endl;

    return 0;
}
