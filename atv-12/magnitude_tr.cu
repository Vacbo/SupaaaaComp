#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <cmath>
#include <iostream>
#include <chrono>

// Functor to square the values
struct square {
    __host__ __device__
    double operator()(const double& x) const {
        return x * x;
    }
};

int main() {
    const int n = 10000000;
    double value = 0.0;

    // Host vector
    thrust::host_vector<double> google_host(n);
    for (int i = 0; i < n; i++) {
        std::cin >> value;
        google_host[i] = value;
    }

    // Transfer to device
    thrust::device_vector<double> google_device = google_host;

    auto start = std::chrono::high_resolution_clock::now();

    // Use transform_reduce to calculate the sum of squares
    double sum_of_squares = thrust::transform_reduce(google_device.begin(), google_device.end(), square(), 0.0, thrust::plus<double>());

    // Compute magnitude as the square root of the sum of squares
    double magnitude = std::sqrt(sum_of_squares);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_time = end - start;

    std::cout << "Magnitude (using transform_reduce): " << magnitude << std::endl;
    std::cout << "Elapsed time: " << elapsed_time.count() << " seconds" << std::endl;

    return 0;
}
