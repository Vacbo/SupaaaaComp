#include <iostream>
#include <vector>
#include <fstream>
#include <chrono>
#include <iomanip>
#include <hip/hip_runtime.h>

using namespace std;
using namespace std::chrono;

// CUDA error checking
#define CHECK_CUDA(call) do { \
    hipError_t e = (call); \
    if(e != hipSuccess) { \
        cerr << "CUDA Error: " << hipGetErrorString(e) << " at " << __FILE__ << ":" << __LINE__ << "\n"; \
        exit(EXIT_FAILURE); \
    } \
} while(0)

// CUDA kernel to check if a vertex i is connected to all vertices in currentClique
__global__ void canAddVertexKernel(const int* d_graph, int numVertices,
                                   const int* d_currentClique, int cliqueSize, int newVertex,
                                   int* d_result) {
    // Each thread checks one vertex in the currentClique
    int idx = threadIdx.x;
    if (idx < cliqueSize) {
        int v = d_currentClique[idx];
        // d_graph[u * numVertices + v] indicates connectivity between u and v
        // Check if newVertex is connected to currentClique[idx]
        if (d_graph[newVertex * numVertices + v] == 0) {
            // If not connected, mark result as 0
            atomicExch(d_result, 0);
        }
    }
}

// Helper function that launches the kernel and returns boolean result
bool canAddVertex(const int* d_graph, int numVertices,
                  const vector<int>& currentClique, int newVertex) {
    int cliqueSize = (int)currentClique.size();
    if (cliqueSize == 0) return true; // If no vertices in clique, always can add

    // Copy currentClique to device
    int* d_currentClique;
    CHECK_CUDA(hipMalloc((void**)&d_currentClique, cliqueSize * sizeof(int)));
    CHECK_CUDA(hipMemcpy(d_currentClique, currentClique.data(), cliqueSize * sizeof(int), hipMemcpyHostToDevice));

    // result on device
    int* d_result;
    CHECK_CUDA(hipMalloc((void**)&d_result, sizeof(int)));
    int initial_result = 1;
    CHECK_CUDA(hipMemcpy(d_result, &initial_result, sizeof(int), hipMemcpyHostToDevice));

    // Launch kernel: one block, cliqueSize threads
    canAddVertexKernel<<<1, cliqueSize>>>(d_graph, numVertices, d_currentClique, cliqueSize, newVertex, d_result);
    CHECK_CUDA(hipDeviceSynchronize());

    // Copy result back
    int host_result;
    CHECK_CUDA(hipMemcpy(&host_result, d_result, sizeof(int), hipMemcpyDeviceToHost));

    // Free device memory
    CHECK_CUDA(hipFree(d_currentClique));
    CHECK_CUDA(hipFree(d_result));

    return (host_result == 1);
}

vector<vector<int>> readGraph(const string& filename, int& numVertices) {
    ifstream file(filename);
    if (!file) {
        cerr << "Error: Cannot open the file at " << filename << endl;
        exit(1);
    }

    int numEdges;
    file >> numVertices >> numEdges;

    vector<vector<int>> graph(numVertices, vector<int>(numVertices, 0));
    for (int i = 0; i < numEdges; ++i) {
        int u, v;
        file >> u >> v;
        u -= 1;
        v -= 1;
        graph[u][v] = 1;
        graph[v][u] = 1;
    }

    return graph;
}

// Exhaustive clique search with pruning, using GPU to check connectivity.
void findCliquesGPU(const int* d_graph, int numVertices, vector<int>& currentClique,
                    vector<int>& maxClique, int start) {
    // Pruning
    if ((int)currentClique.size() > (int)maxClique.size()) {
        maxClique = currentClique;
    }

    // Another simple pruning:
    // If even adding all remaining vertices won't exceed the maxClique size, stop
    if ((int)currentClique.size() + (numVertices - start) <= (int)maxClique.size()) {
        return;
    }

    for (int i = start; i < numVertices; ++i) {
        if ((int)currentClique.size() + (numVertices - i) <= (int)maxClique.size()) {
            break; // no chance of improvement
        }

        // Use GPU to check if we can add vertex i
        if (canAddVertex(d_graph, numVertices, currentClique, i)) {
            currentClique.push_back(i);
            findCliquesGPU(d_graph, numVertices, currentClique, maxClique, i + 1);
            currentClique.pop_back();
        }
    }
}

int main(int argc, char** argv) {
    if (argc < 2) {
        cerr << "Usage: " << argv[0] << " <path_to_graph_file>" << endl;
        return 1;
    }

    string filename = argv[1];
    int numVertices;
    vector<vector<int>> graph = readGraph(filename, numVertices);

    // Flatten the graph for GPU
    vector<int> flatGraph(numVertices * numVertices);
    for (int i = 0; i < numVertices; ++i) {
        for (int j = 0; j < numVertices; ++j) {
            flatGraph[i * numVertices + j] = graph[i][j];
        }
    }

    int* d_graph;
    CHECK_CUDA(hipMalloc((void**)&d_graph, numVertices * numVertices * sizeof(int)));
    CHECK_CUDA(hipMemcpy(d_graph, flatGraph.data(), numVertices * numVertices * sizeof(int), hipMemcpyHostToDevice));

    auto startTime = high_resolution_clock::now();

    vector<int> maxClique;
    vector<int> currentClique;

    // Start exhaustive search
    findCliquesGPU(d_graph, numVertices, currentClique, maxClique, 0);

    auto endTime = high_resolution_clock::now();
    duration<double> duration = endTime - startTime;

    cout << "Maximum Clique Size: " << maxClique.size() << endl;
    cout << "Maximum Clique Nodes: ";
    for (int v : maxClique) {
        cout << (v + 1) << " ";
    }
    cout << endl;
    cout << "Time taken: " << fixed << setprecision(6) << duration.count() << " seconds" << endl;

    CHECK_CUDA(hipFree(d_graph));

    return 0;
}
