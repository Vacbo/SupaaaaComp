#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/extrema.h>
#include <thrust/reduce.h>
#include <iostream>
using namespace std;

int main() {
    int n = 3650;  // Assume 10 years of data (10 * 365 = 3650)
    double value = 0.0;

    thrust::host_vector<double> host(n, 0);
    for (int i = 0; i < n; i++) {
        cin >> value;              // Lê um valor da entrada padrão
        host[i] = value;           // Armazena o valor lido no vetor host
    }

    thrust::device_vector<double> dev(host);  // Copia dados para o device

    // Total period statistics
    double total_avg = thrust::reduce(dev.begin(), dev.end(), 0.0) / n;
    auto total_minmax = thrust::minmax_element(dev.begin(), dev.end());

    // Last year statistics (last 365 days)
    int last_year_idx = n - 365;
    double last_year_avg = thrust::reduce(dev.begin() + last_year_idx, dev.end(), 0.0) / 365;
    auto last_year_minmax = thrust::minmax_element(dev.begin() + last_year_idx, dev.end());

    // Print results
    cout << "Total period avg price: " << total_avg << endl;
    cout << "Total period min price: " << *(total_minmax.first) << endl;
    cout << "Total period max price: " << *(total_minmax.second) << endl;

    cout << "Last year avg price: " << last_year_avg << endl;
    cout << "Last year min price: " << *(last_year_minmax.first) << endl;
    cout << "Last year max price: " << *(last_year_minmax.second) << endl;

    return 0;
}
